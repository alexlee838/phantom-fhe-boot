#include "hip/hip_runtime.h"
#include "ckks.h"
#include "fft.h"

using namespace std;
using namespace phantom;
using namespace phantom::util;
using namespace phantom::arith;

__global__ void bit_reverse_kernel(hipDoubleComplex *dst, hipDoubleComplex *src, uint64_t in_size,
                                   uint32_t log_n)
{
    for (uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
         tid < in_size; tid += blockDim.x * gridDim.x)
    {
        dst[reverse_bits_uint32(tid, log_n)] = src[tid];
    }
}

__global__ void extend_sparse_ckks(uint64_t *out, const uint64_t *in,
                                   int val_size, int slots, size_t total_size)
{
    size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    size_t stride = slots / val_size;

    if (tid < total_size)
    {
        int target_idx = tid * stride;
        out[target_idx] = in[tid];
    }
}

__global__ void shrink_sparse_ckks(uint64_t *out, const uint64_t *in,
                                   int val_size, int slots, size_t total_size)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = slots / val_size;

    if (tid < total_size)
    {
        int source_idx = tid * stride;
        out[tid] = in[source_idx];
    }
}

PhantomCKKSEncoder::PhantomCKKSEncoder(const PhantomContext &context)
{
    const auto &s = hipStreamPerThread;

    auto &context_data = context.get_context_data(first_chain_index_);
    auto &parms = context_data.parms();
    std::size_t coeff_count = parms.poly_modulus_degree();

    if (parms.scheme() != scheme_type::ckks)
    {
        throw std::invalid_argument("unsupported scheme");
    }

    slots_ = coeff_count >> 1;
    uint32_t m = coeff_count << 1;
    uint32_t slots_half = slots_ >> 1;

    gpu_ckks_msg_vec_ = std::make_unique<DCKKSEncoderInfo>(coeff_count, s);

    // We need m powers of the primitive 2n-th root, m = 2n
    root_powers_.reserve(m);
    rotation_group_.reserve(slots_half);

    uint32_t gen = 5;
    uint32_t pos = 1; // Position in normal bit order
    for (size_t i = 0; i < slots_half; i++)
    {
        // Set the bit-reversed locations
        rotation_group_[i] = pos;

        // Next primitive root
        pos *= gen; // 5^i mod m
        pos &= (m - 1);
    }

    // Powers of the primitive 2n-th root have 4-fold symmetry
    if (m >= 8)
    {
        complex_roots_ = std::make_unique<util::ComplexRoots>(util::ComplexRoots(static_cast<size_t>(m)));
        for (size_t i = 0; i < m; i++)
        {
            root_powers_[i] = complex_roots_->get_root(i);
        }
    }
    else if (m == 4)
    {
        root_powers_[0] = {1, 0};
        root_powers_[1] = {0, 1};
        root_powers_[2] = {-1, 0};
        root_powers_[3] = {0, -1};
    }

    hipMemcpyAsync(gpu_ckks_msg_vec_->twiddle(), root_powers_.data(), m * sizeof(hipDoubleComplex),
                    hipMemcpyHostToDevice, s);
    hipMemcpyAsync(gpu_ckks_msg_vec_->mul_group(), rotation_group_.data(), slots_half * sizeof(uint32_t),
                    hipMemcpyHostToDevice, s);
}

void PhantomCKKSEncoder::encode_internal(const PhantomContext &context, const std::vector<hipDoubleComplex> &values,
                                         size_t chain_index, double scale,
                                         PhantomPlaintext &destination, const hipStream_t &stream)
{
    auto &context_data = context.get_context_data(chain_index);
    auto &parms = context_data.parms();
    auto &coeff_modulus = parms.coeff_modulus();
    auto &rns_tool = context_data.gpu_rns_tool();
    size_t coeff_modulus_size = coeff_modulus.size();
    size_t coeff_count = parms.poly_modulus_degree();
    size_t log_slot_count = arith::get_power_of_two(slots_);
    size_t values_size = values.size();

    if (values.empty())
    {
        throw std::invalid_argument("Input vector is empty");
    }
    else if (values_size > slots_)
    {
        throw std::invalid_argument("Input vector exceeds max slots");
    }

    // Check that scale is positive and not too large
    if (scale <= 0 || (static_cast<int>(log2(scale)) + 1 >= context_data.total_coeff_modulus_bit_count()))
    {
        throw std::invalid_argument("scale out of bounds");
    }

    auto temp = make_cuda_auto_ptr<hipDoubleComplex>(values_size, stream);
    hipMemcpyAsync(temp.get(), values.data(), values_size * sizeof(hipDoubleComplex), hipMemcpyHostToDevice,
                    stream);

    hipMemsetAsync(gpu_ckks_msg_vec_->in(), 0, slots_ * sizeof(hipDoubleComplex), stream);

    size_t gridDimGlb = std::ceil((float)values_size / (float)blockDimGlb.x);
    bit_reverse_kernel<<<gridDimGlb, blockDimGlb, 0, stream>>>(
        gpu_ckks_msg_vec_->in(), temp.get(), values_size, log_slot_count);

    double fix = scale / static_cast<double>(slots_);

    special_fft_backward(*gpu_ckks_msg_vec_, log_slot_count, fix, stream);

    // TODO: boundary check on GPU
    vector<hipDoubleComplex> temp2(slots_);
    hipMemcpyAsync(temp2.data(), gpu_ckks_msg_vec_->in(), slots_ * sizeof(hipDoubleComplex),
                    hipMemcpyDeviceToHost, stream);
    // explicit stream synchronize to avoid error
    hipStreamSynchronize(stream);

    double max_coeff = 0;
    for (std::size_t i = 0; i < slots_; i++)
    {
        max_coeff = std::max(max_coeff, std::fabs(temp2[i].x));
    }
    for (std::size_t i = 0; i < slots_; i++)
    {
        max_coeff = std::max(max_coeff, std::fabs(temp2[i].y));
    }
    // Verify that the values are not too large to fit in coeff_modulus
    // Note that we have an extra + 1 for the sign bit
    // Don't compute logarithmis of numbers less than 1
    int max_coeff_bit_count = static_cast<int>(std::ceil(std::log2(std::max(max_coeff, 1.0)))) + 1;

    if (max_coeff_bit_count >= context_data.total_coeff_modulus_bit_count())
    {
        throw std::invalid_argument("encoded values are too large");
    }

    rns_tool.base_Ql().decompose_array(destination.data(), gpu_ckks_msg_vec_->in(), coeff_count, max_coeff_bit_count,
                                       stream);

    nwt_2d_radix8_forward_inplace(destination.data(), context.gpu_rns_tables(), coeff_modulus_size, 0, stream);

    destination.chain_index_ = chain_index;
    destination.scale_ = scale;
}

void PhantomCKKSEncoder::encode_sparse_internal(const PhantomContext &context, const std::vector<hipDoubleComplex> &values,
                                                size_t chain_index, double scale, PhantomPlaintext &destination, const hipStream_t &stream)
{
    auto &context_data = context.get_context_data(chain_index);
    auto &parms = context_data.parms();
    auto &coeff_modulus = parms.coeff_modulus();
    auto &rns_tool = context_data.gpu_rns_tool();
    size_t coeff_modulus_size = coeff_modulus.size();
    size_t coeff_count = parms.poly_modulus_degree();
    size_t log_slot_count = arith::get_power_of_two(slots_);
    size_t values_size = values.size();
    size_t log_val_size_count = arith::get_power_of_two(values_size);
    
    const auto &s = hipStreamPerThread;
    if (values.empty())
    {
        throw std::invalid_argument("Input vector is empty");
    }
    else if (values_size > slots_)
    {
        throw std::invalid_argument("Input vector exceeds max slots");
    }

    // Check that scale is positive and not too large
    if (scale <= 0 || (static_cast<int>(log2(scale)) + 1 >= context_data.total_coeff_modulus_bit_count()))
    {
        throw std::invalid_argument("scale out of bounds");
    }

    if(!sparse_context_)
    {
        throw std::invalid_argument("Sparse context is not initialized");
    }

    auto &sparse_rns_tool = sparse_context_->get_context_data(chain_index).gpu_rns_tool();


    auto temp = make_cuda_auto_ptr<hipDoubleComplex>(values_size, stream);

    hipMemcpyAsync(temp.get(), values.data(), values_size * sizeof(hipDoubleComplex), hipMemcpyHostToDevice, stream);

    hipMemsetAsync(sparse_gpu_ckks_msg_vec_->in(), 0, values_size * sizeof(hipDoubleComplex), stream);

    size_t gridDimGlb = std::ceil((float)values_size / (float)blockDimGlb.x);
    bit_reverse_kernel<<<gridDimGlb, blockDimGlb, 0, stream>>>(
        sparse_gpu_ckks_msg_vec_->in(), temp.get(), values_size, log_val_size_count);

    double fix = scale / static_cast<double>(values_size);

    special_fft_backward(*sparse_gpu_ckks_msg_vec_, log_val_size_count, fix, stream);

    // TODO: boundary check on GPU
    vector<hipDoubleComplex> temp2(values_size);
    hipMemcpyAsync(temp2.data(), sparse_gpu_ckks_msg_vec_->in(), values_size * sizeof(hipDoubleComplex),
                    hipMemcpyDeviceToHost, stream);
    // explicit stream synchronize to avoid error
    hipStreamSynchronize(stream);

    double max_coeff = 0;
    for (std::size_t i = 0; i < values_size; i++)
    {
        max_coeff = std::max(max_coeff, std::fabs(temp2[i].x));
    }
    for (std::size_t i = 0; i < values_size; i++)
    {
        max_coeff = std::max(max_coeff, std::fabs(temp2[i].y));
    }
    // Verify that the values are not too large to fit in coeff_modulus
    // Note that we have an extra + 1 for the sign bit
    // Don't compute logarithmis of numbers less than 1
    int max_coeff_bit_count = static_cast<int>(std::ceil(std::log2(std::max(max_coeff, 1.0)))) + 1;

    if (max_coeff_bit_count >= context_data.total_coeff_modulus_bit_count())
    {
        throw std::invalid_argument("encoded values are too large");
    }

    PhantomPlaintext sparse_destination;
    sparse_destination.resize(coeff_modulus_size, values_size * 2, s);

    sparse_rns_tool.base_Ql().decompose_array(sparse_destination.data(), sparse_gpu_ckks_msg_vec_->in(), values_size * 2, max_coeff_bit_count,
                                              stream);

    hipMemsetAsync(destination.data(), 0, coeff_count * coeff_modulus_size * sizeof(uint64_t), stream);

    size_t total_size = values_size * 2 * coeff_modulus_size;
    gridDimGlb = std::ceil((float)(total_size) / (float)blockDimGlb.x);

    extend_sparse_ckks<<<gridDimGlb, blockDimGlb, 0, stream>>>(destination.data(), sparse_destination.data(), values_size, slots_, total_size);

    nwt_2d_radix8_forward_inplace(destination.data(), context.gpu_rns_tables(), coeff_modulus_size, 0, stream);

    destination.chain_index_ = chain_index;
    destination.scale_ = scale;
}

void PhantomCKKSEncoder::encode_internal_ext(const PhantomContext &context, const std::vector<hipDoubleComplex> &values,
                                             size_t chain_index, double scale,
                                             PhantomPlaintext &destination, const hipStream_t &stream)
{

    auto &context_data = context.get_context_data(chain_index);
    auto &parms = context_data.parms();
    auto &coeff_modulus = parms.coeff_modulus();
    auto &rns_tool = context_data.gpu_rns_tool();
    size_t size_Ql = coeff_modulus.size();
    size_t size_Q = context.get_context_data(context.get_first_index()).parms().coeff_modulus().size();
    size_t size_P = context.get_context_data(0).parms().special_modulus_size();
    size_t size_QlP = size_Ql + size_P;
    size_t size_QP = size_Q + size_P;
    size_t coeff_count = parms.poly_modulus_degree();
    size_t log_slot_count = arith::get_power_of_two(slots_);
    size_t values_size = values.size();

    if (values.empty())
    {
        throw std::invalid_argument("Input vector is empty");
    }
    else if (values_size > slots_)
    {
        throw std::invalid_argument("Input vector exceeds max slots");
    }

    // Check that scale is positive and not too large
    if (scale <= 0 || (static_cast<int>(log2(scale)) + 1 >= context_data.total_coeff_modulus_bit_count()))
    {
        throw std::invalid_argument("scale out of bounds");
    }

    auto temp = make_cuda_auto_ptr<hipDoubleComplex>(values_size, stream);
    hipMemcpyAsync(temp.get(), values.data(), values_size * sizeof(hipDoubleComplex), hipMemcpyHostToDevice, stream);

    hipMemsetAsync(gpu_ckks_msg_vec_->in(), 0, slots_ * sizeof(hipDoubleComplex), stream);

    size_t gridDimGlb = std::ceil((float)values_size / (float)blockDimGlb.x);
    bit_reverse_kernel<<<gridDimGlb, blockDimGlb, 0, stream>>>(
        gpu_ckks_msg_vec_->in(), temp.get(), values_size, log_slot_count);

    double fix = scale / static_cast<double>(slots_);

    special_fft_backward(*gpu_ckks_msg_vec_, log_slot_count, fix, stream);

    // TODO: boundary check on GPU
    vector<hipDoubleComplex> temp2(slots_);
    hipMemcpyAsync(temp2.data(), gpu_ckks_msg_vec_->in(), slots_ * sizeof(hipDoubleComplex),
                    hipMemcpyDeviceToHost, stream);
    // explicit stream synchronize to avoid error
    hipStreamSynchronize(stream);

    double max_coeff = 0;
    for (std::size_t i = 0; i < slots_; i++)
    {
        max_coeff = std::max(max_coeff, std::fabs(temp2[i].x));
    }
    for (std::size_t i = 0; i < slots_; i++)
    {
        max_coeff = std::max(max_coeff, std::fabs(temp2[i].y));
    }
    // Verify that the values are not too large to fit in coeff_modulus
    // Note that we have an extra + 1 for the sign bit
    // Don't compute logarithmis of numbers less than 1
    int max_coeff_bit_count = static_cast<int>(std::ceil(std::log2(std::max(max_coeff, 1.0)))) + 1;

    if (max_coeff_bit_count >= context_data.total_coeff_modulus_bit_count())
    {
        throw std::invalid_argument("encoded values are too large");
    }

    rns_tool.base_QlP().decompose_array(destination.data(), gpu_ckks_msg_vec_->in(), coeff_count, max_coeff_bit_count, stream);

    nwt_2d_radix8_forward_inplace_include_special_mod(destination.data(), context.gpu_rns_tables(), size_QlP, 0, size_QP, size_P, stream);
    destination.chain_index_ = chain_index;
    destination.scale_ = scale;
}

void PhantomCKKSEncoder::encode_sparse_internal_ext(const PhantomContext &context, const std::vector<hipDoubleComplex> &values,
                                                    size_t chain_index, double scale, PhantomPlaintext &destination, const hipStream_t &stream)
{
    // Needs to be Debugged
    auto &context_data = context.get_context_data(chain_index);
    auto &parms = context_data.parms();
    auto &coeff_modulus = parms.coeff_modulus();
    auto &rns_tool = context_data.gpu_rns_tool();
    size_t size_Ql = coeff_modulus.size();
    size_t size_Q = context.get_context_data(context.get_first_index()).parms().coeff_modulus().size();
    size_t size_P = context.get_context_data(0).parms().special_modulus_size();
    size_t size_QlP = size_Ql + size_P;
    size_t size_QP = size_Q + size_P;
    size_t coeff_count = parms.poly_modulus_degree();
    size_t log_slot_count = arith::get_power_of_two(slots_);
    size_t values_size = values.size();
    size_t log_val_size_count = arith::get_power_of_two(values_size);

    if (!sparse_bootstrap_context_)
    {
        throw std::invalid_argument("Sparse bootstrap context is not initialized");
    }

    auto &sparse_rns_tool = sparse_bootstrap_context_->get_context_data(chain_index).gpu_rns_tool();

    const auto &s = hipStreamPerThread;

    if (values.empty())
    {
        throw std::invalid_argument("Input vector is empty");
    }
    else if (values_size > slots_)
    {
        throw std::invalid_argument("Input vector exceeds max slots");
    }

    // Check that scale is positive and not too large
    if (scale <= 0 || (static_cast<int>(log2(scale)) + 1 >= context_data.total_coeff_modulus_bit_count()))
    {
        throw std::invalid_argument("scale out of bounds");
    }

    auto temp = make_cuda_auto_ptr<hipDoubleComplex>(values_size, stream);

    hipMemcpyAsync(temp.get(), values.data(), values_size * sizeof(hipDoubleComplex), hipMemcpyHostToDevice, stream);

    hipMemsetAsync(sparse_bootstrap_gpu_ckks_msg_vec_->in(), 0, values_size * sizeof(hipDoubleComplex), stream);

    size_t gridDimGlb = std::ceil((float)values_size / (float)blockDimGlb.x);
    bit_reverse_kernel<<<gridDimGlb, blockDimGlb, 0, stream>>>(
        sparse_bootstrap_gpu_ckks_msg_vec_->in(), temp.get(), values_size, log_val_size_count);

    double fix = scale / static_cast<double>(values_size);

    special_fft_backward(*sparse_bootstrap_gpu_ckks_msg_vec_, log_val_size_count, fix, stream);

    // TODO: boundary check on GPU
    vector<hipDoubleComplex> temp2(values_size);
    hipMemcpyAsync(temp2.data(), sparse_bootstrap_gpu_ckks_msg_vec_->in(), values_size * sizeof(hipDoubleComplex),
                    hipMemcpyDeviceToHost, stream);
    // explicit stream synchronize to avoid error
    hipStreamSynchronize(stream);

    double max_coeff = 0;
    for (std::size_t i = 0; i < values_size; i++)
    {
        max_coeff = std::max(max_coeff, std::fabs(temp2[i].x));
    }
    for (std::size_t i = 0; i < values_size; i++)
    {
        max_coeff = std::max(max_coeff, std::fabs(temp2[i].y));
    }
    // Verify that the values are not too large to fit in coeff_modulus
    // Note that we have an extra + 1 for the sign bit
    // Don't compute logarithmis of numbers less than 1
    int max_coeff_bit_count = static_cast<int>(std::ceil(std::log2(std::max(max_coeff, 1.0)))) + 1;

    if (max_coeff_bit_count >= context_data.total_coeff_modulus_bit_count())
    {
        throw std::invalid_argument("encoded values are too large");
    }

    PhantomPlaintext sparse_destination;
    sparse_destination.resize(size_QlP, values_size * 2, s);

    sparse_rns_tool.base_QlP().decompose_array(sparse_destination.data(), sparse_bootstrap_gpu_ckks_msg_vec_->in(), values_size * 2, max_coeff_bit_count,
                                               stream);

    hipMemsetAsync(destination.data(), 0, coeff_count * size_QlP * sizeof(uint64_t), stream);

    size_t total_size = values_size * 2 * size_QlP;
    gridDimGlb = std::ceil((float)(total_size) / (float)blockDimGlb.x);

    extend_sparse_ckks<<<gridDimGlb, blockDimGlb, 0, stream>>>(destination.data(), sparse_destination.data(), values_size, slots_, total_size);

    nwt_2d_radix8_forward_inplace_include_special_mod(destination.data(), context.gpu_rns_tables(), size_QlP, 0, size_QP, size_P, stream);

    destination.chain_index_ = chain_index;
    destination.scale_ = scale;
}

void PhantomCKKSEncoder::decode_internal(const PhantomContext &context, const PhantomPlaintext &plain,
                                         std::vector<hipDoubleComplex> &destination, const hipStream_t &stream)
{
    auto &context_data = context.get_context_data(plain.chain_index_);
    auto &parms = context_data.parms();
    auto &coeff_modulus = parms.coeff_modulus();
    auto &rns_tool = context_data.gpu_rns_tool();
    const size_t coeff_modulus_size = coeff_modulus.size();
    const size_t coeff_count = parms.poly_modulus_degree();
    size_t log_slot_count = arith::get_power_of_two(slots_);
    const size_t rns_poly_uint64_count = coeff_count * coeff_modulus_size;

    if (plain.scale() <= 0 ||
        (static_cast<int>(log2(plain.scale())) >= context_data.total_coeff_modulus_bit_count()))
    {
        throw std::invalid_argument("scale out of bounds");
    }

    auto upper_half_threshold = context_data.upper_half_threshold();
    int logn = arith::get_power_of_two(coeff_count);
    auto gpu_upper_half_threshold = make_cuda_auto_ptr<uint64_t>(upper_half_threshold.size(), stream);
    hipMemcpyAsync(gpu_upper_half_threshold.get(), upper_half_threshold.data(),
                    upper_half_threshold.size() * sizeof(uint64_t), hipMemcpyHostToDevice, stream);

    hipMemsetAsync(gpu_ckks_msg_vec_->in(), 0, slots_ * sizeof(hipDoubleComplex), stream);

    // Quick sanity check
    if ((logn < 0) || (coeff_count < POLY_MOD_DEGREE_MIN) || (coeff_count > POLY_MOD_DEGREE_MAX))
    {
        throw std::logic_error("invalid parameters");
    }

    double inv_scale = double(1.0) / plain.scale();
    // Create mutable copy of input
    auto plain_copy = make_cuda_auto_ptr<uint64_t>(rns_poly_uint64_count, stream);
    hipMemcpyAsync(plain_copy.get(), plain.data(), rns_poly_uint64_count * sizeof(uint64_t), hipMemcpyDeviceToDevice,
                    stream);

    nwt_2d_radix8_backward_inplace(plain_copy.get(), context.gpu_rns_tables(), coeff_modulus_size, 0, stream);

    // CRT-compose the polynomial
    if (plain.chain_index_ != 0)
    {
        rns_tool.base_Ql().compose_array(gpu_ckks_msg_vec().in(), plain_copy.get(), gpu_upper_half_threshold.get(),
                                         inv_scale, coeff_count, stream);
    }

    else
    {
        rns_tool.base_QlP().compose_array(gpu_ckks_msg_vec().in(), plain_copy.get(), gpu_upper_half_threshold.get(),
                                          inv_scale, coeff_count, stream);
    }

    special_fft_forward(*gpu_ckks_msg_vec_, log_slot_count, stream);

    auto out = make_cuda_auto_ptr<hipDoubleComplex>(slots_, stream);
    size_t gridDimGlb = std::ceil((float)slots_ / (float)blockDimGlb.x);
    bit_reverse_kernel<<<gridDimGlb, blockDimGlb, 0, stream>>>(
        out.get(), gpu_ckks_msg_vec_->in(), slots_, log_slot_count);

    destination.resize(slots_);
    hipMemcpyAsync(destination.data(), out.get(), slots_ * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost, stream);

    // explicit synchronization in case user wants to use the result immediately
    hipStreamSynchronize(stream);
}

void PhantomCKKSEncoder::decode_sparse_internal(const PhantomContext &context, const PhantomPlaintext &plain, size_t val_size,
                                                std::vector<hipDoubleComplex> &destination, const hipStream_t &stream)
{
    // Needs to be Debugged
    auto &context_data = context.get_context_data(plain.chain_index_);
    if (!sparse_context_)
    {
        throw std::invalid_argument("Sparse  context is not initialized");
    }

    auto &sparse_parms = sparse_context_->get_context_data(plain.chain_index_).parms();
    auto &sparse_coeff_modulus = sparse_parms.coeff_modulus();
    auto &sparse_rns_tool = context_data.gpu_rns_tool();
    const size_t sparse_coeff_modulus_size = sparse_coeff_modulus.size();
    const size_t sparse_coeff_count = sparse_parms.poly_modulus_degree();
    size_t log_slot_count = arith::get_power_of_two(slots_);

    auto &parms = context_data.parms();
    auto &coeff_modulus = parms.coeff_modulus();
    auto &rns_tool = context_data.gpu_rns_tool();
    size_t coeff_modulus_size = coeff_modulus.size();
    size_t coeff_count = parms.poly_modulus_degree();

    const size_t rns_poly_uint64_count = coeff_count * coeff_modulus_size;

    size_t log_val_size_count = arith::get_power_of_two(val_size);

    if (plain.scale() <= 0 ||
        (static_cast<int>(log2(plain.scale())) >= context_data.total_coeff_modulus_bit_count()))
    {
        throw std::invalid_argument("scale out of bounds");
    }

    // Quick sanity check
    int logn = arith::get_power_of_two(coeff_count);
    if ((logn < 0) || (coeff_count < POLY_MOD_DEGREE_MIN) || (coeff_count > POLY_MOD_DEGREE_MAX))
    {
        throw std::logic_error("invalid parameters");
    }

    auto sparse_upper_half_threshold = sparse_context_->get_context_data(plain.chain_index_).upper_half_threshold();
    auto sparse_gpu_upper_half_threshold = make_cuda_auto_ptr<uint64_t>(sparse_upper_half_threshold.size(), stream);
    hipMemcpyAsync(sparse_gpu_upper_half_threshold.get(), sparse_upper_half_threshold.data(),
                    sparse_upper_half_threshold.size() * sizeof(uint64_t), hipMemcpyHostToDevice, stream);

    hipMemsetAsync(sparse_gpu_ckks_msg_vec_->in(), 0, val_size * sizeof(hipDoubleComplex), stream);

    double inv_scale = double(1.0) / plain.scale();
    // Create mutable copy of input

    size_t total_size = val_size * 2 * sparse_coeff_modulus_size;

    auto plain_copy = make_cuda_auto_ptr<uint64_t>(rns_poly_uint64_count, stream);
    auto sparse_plain = make_cuda_auto_ptr<uint64_t>(total_size, stream);

    hipMemcpyAsync(plain_copy.get(), plain.data(), rns_poly_uint64_count * sizeof(uint64_t), hipMemcpyDeviceToDevice,
                    stream);

    nwt_2d_radix8_backward_inplace(plain_copy.get(), context.gpu_rns_tables(), coeff_modulus_size, 0, stream);

    size_t gridDimGlb = std::ceil((float)total_size / (float)blockDimGlb.x);
    shrink_sparse_ckks<<<gridDimGlb, blockDimGlb, 0, stream>>>(sparse_plain.get(), plain_copy.get(), val_size, slots_, total_size);

    // CRT-compose the polynomial
    if (plain.chain_index_ != 0)
    {
        sparse_rns_tool.base_Ql().compose_array(sparse_gpu_ckks_msg_vec_->in(), sparse_plain.get(), sparse_gpu_upper_half_threshold.get(),
                                                inv_scale, sparse_coeff_count, stream);
    }

    else
    {
        sparse_rns_tool.base_QlP().compose_array(sparse_gpu_ckks_msg_vec_->in(), sparse_plain.get(), sparse_gpu_upper_half_threshold.get(),
                                                 inv_scale, sparse_coeff_count, stream);
    }
    gridDimGlb = std::ceil((float)val_size / (float)blockDimGlb.x);

    special_fft_forward(*sparse_gpu_ckks_msg_vec_, log_val_size_count, stream);

    auto out = make_cuda_auto_ptr<hipDoubleComplex>(val_size, stream);

    bit_reverse_kernel<<<gridDimGlb, blockDimGlb, 0, stream>>>(
        out.get(), sparse_gpu_ckks_msg_vec_->in(), val_size, log_val_size_count);

    destination.resize(val_size);
    hipMemcpyAsync(destination.data(), out.get(), val_size * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost, stream);

    // explicit synchronization in case user wants to use the result immediately
    hipStreamSynchronize(stream);
}

void PhantomCKKSEncoder::decode_internal_ext(const PhantomContext &context, const PhantomPlaintext &plain,
                                             std::vector<hipDoubleComplex> &destination, const hipStream_t &stream)
{
    auto &context_data = context.get_context_data(plain.chain_index_);
    auto &parms = context_data.parms();
    auto &coeff_modulus = parms.coeff_modulus();
    auto &rns_tool = context_data.gpu_rns_tool();
    const size_t coeff_modulus_size = coeff_modulus.size();
    const size_t coeff_count = parms.poly_modulus_degree();
    size_t log_slot_count = arith::get_power_of_two(slots_);
    const size_t rns_poly_uint64_count = coeff_count * coeff_modulus_size;

    if (plain.scale() <= 0 ||
        (static_cast<int>(log2(plain.scale())) >= context_data.total_coeff_modulus_bit_count()))
    {
        throw std::invalid_argument("scale out of bounds");
    }

    auto upper_half_threshold = context_data.upper_half_threshold();
    int logn = arith::get_power_of_two(coeff_count);
    auto gpu_upper_half_threshold = make_cuda_auto_ptr<uint64_t>(upper_half_threshold.size(), stream);
    hipMemcpyAsync(gpu_upper_half_threshold.get(), upper_half_threshold.data(),
                    upper_half_threshold.size() * sizeof(uint64_t), hipMemcpyHostToDevice, stream);

    hipMemsetAsync(gpu_ckks_msg_vec_->in(), 0, slots_ * sizeof(hipDoubleComplex), stream);

    // Quick sanity check
    if ((logn < 0) || (coeff_count < POLY_MOD_DEGREE_MIN) || (coeff_count > POLY_MOD_DEGREE_MAX))
    {
        throw std::logic_error("invalid parameters");
    }

    double inv_scale = double(1.0) / plain.scale();
    // Create mutable copy of input
    auto plain_copy = make_cuda_auto_ptr<uint64_t>(rns_poly_uint64_count, stream);
    hipMemcpyAsync(plain_copy.get(), plain.data(), rns_poly_uint64_count * sizeof(uint64_t), hipMemcpyDeviceToDevice,
                    stream);

    nwt_2d_radix8_backward_inplace(plain_copy.get(), context.gpu_rns_tables(), coeff_modulus_size, 0, stream);

    rns_tool.base_QlP().compose_array(gpu_ckks_msg_vec().in(), plain_copy.get(), gpu_upper_half_threshold.get(),
                                      inv_scale, coeff_count, stream);

    special_fft_forward(*gpu_ckks_msg_vec_, log_slot_count, stream);

    auto out = make_cuda_auto_ptr<hipDoubleComplex>(slots_, stream);
    size_t gridDimGlb = std::ceil((float)slots_ / (float)blockDimGlb.x);
    bit_reverse_kernel<<<gridDimGlb, blockDimGlb, 0, stream>>>(
        out.get(), gpu_ckks_msg_vec_->in(), slots_, log_slot_count);

    destination.resize(slots_);
    hipMemcpyAsync(destination.data(), out.get(), slots_ * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost, stream);

    // explicit synchronization in case user wants to use the result immediately
    hipStreamSynchronize(stream);
}
